#include "hip/hip_runtime.h"
#include <stdio.h>

#define ARRAYSIZE(x)  (sizeof(x)/sizeof(*(x)))



int attributes[683][9];
int classes[683][2];
double node1weights[500][9],node2weights[500][500],node3weights[500][500],outputlayer[2][500];



double errRate = 0.01;


__global__ void feed_forward_loop1(int training_sample,double *forpassl1,double node1weights[500][9],int attributes[683][9])
{

for(int i=0;i<500;i++){
			forpassl1[i]=0;
			double multifir = 0;
			for(int j=0;j<9;j++){
				multifir=(multifir)+node1weights[i][j]*(attributes[training_sample][j]);
			}
			int e=-4;int ee=e;
			double T = 1;if(e<0){ee=-e;}
			for(int k=1;k<=ee;k++)T = T*multifir;
			T = (e<0) ? 1/T : T;
			forpassl1[i]=1/(1+T);
		}

}
__global__ void feed_forward_loop2(int training_sample,double *forpassl2,double node2weights[500][500],double *forpassl1)
{
for(int i=0;i<500;i++){
			forpassl2[i]=0;
			int multifir = 0;
			for(int j=0;j<500;j++){
				multifir=multifir+node2weights[i][j]*forpassl1[j];
			}
			int e=-4;int ee=e;
                        double T = 1;if(e<0){ee=-e;}
                        for(int k=1;k<=ee;k++)T = T*multifir;
                        T = (e<0) ? 1/T : T;
                        forpassl1[i]=1/(1+T);}
}

__global__ void feed_forward_loop3(int training_sample,double *forpassl3,double node3weights[500][500],double *forpassl2)
{
for(int i=0;i<500;i++){
                        forpassl3[i]=0;
                        int multifir = 0;
                        for(int j=0;j<500;j++){
                                multifir=multifir+node3weights[i][j]*forpassl2[j];
                        }
                        int e=-4;int ee=e;
                        double T = 1;if(e<0){ee=-e;}
                        for(int k=1;k<=ee;k++)T = T*multifir;
                        T = (e<0) ? 1/T : T;
                        forpassl3[i]=1/(1+T);}
}

__global__ void feed_forward_loop4(double *forpassout,double outputlayer[2][500],double *forpassl3)
{

for(int i=0;i<2;i++){
			forpassout[i]=0;
			int multifir = 0;
			for(int j=0;j<500;j++){
				multifir=multifir+outputlayer[i][j]*forpassl3[j];
			}
			
int e=-4;int ee=e;
                        double T = 1;if(e<0){ee=-e;}
                        for(int k=1;k<=ee;k++)T = T*multifir;
                        T = (e<0) ? 1/T : T;
                        forpassout[i]=1/(1+T);
		}
}

__global__ void feed_forward_loop5(int training_sample,double *errout,double *forpassout,int classes[683][2])
{

for(int i=0;i<2;i++){
			errout[i]=forpassout[i]*(1-forpassout[i])*(classes[training_sample][i]-forpassout[i]);
		}
}
__global__ void feed_forward_loop6(double *errout,double outputlayer[2][500],double *errl3,double *forpassl3)
{

for(int i=0;i<500;i++)
		{	int sum=0;
			for(int k=0;k<2;k++)
				{sum=sum+errout[k]*outputlayer[k][i];}

			errl3[i]=forpassl3[i]*(1-forpassl3[i])*sum;
		}
}
__global__ void feed_forward_loop7(double *errl3,double node3weights[500][500],double *forpassl2,double *errl2)
{

for(int i=0;i<500;i++)
		{	int sum=0;
			for(int k=0;k<500;k++)
				{sum=sum+errl3[k]*node3weights[k][i];}

			errl2[i]=forpassl2[i]*(1-forpassl2[i])*sum;
		}

}

__global__ void feed_forward_loop8(double *errl2,double node2weights[500][500],double *forpassl1,double *errl1)
{


for(int i=0;i<500;i++)
		{	int sum=0;
			for(int k=0;k<500;k++)
				{sum=sum+errl2[k]*node2weights[k][i];}

			errl1[i]=forpassl1[i]*(1-forpassl1[i])*sum;
		}

}

__global__ void feed_forward_loop9(double outputlayer[2][500],double errRate,double *errout,double *forpassl3)
{

for(int k=0;k<2;k++)
			for(int j=0;j<500;j++)
			{
				outputlayer[k][j] = outputlayer[k][j] + errRate*(errout[k]*forpassl3[j]);
			}



}

__global__ void feed_forward_loop10(double node3weights[500][500],double *forpassl2,double errRate,double *errl3)
{

for(int k=0;k<500;k++)
			for(int j=0;j<500;j++)
			{
				node3weights[k][j] = node3weights[k][j] + errRate*(errl3[k]*forpassl2[j]);
			}


}

__global__ void feed_forward_loop11(double node2weights[500][500],double *forpassl1,double errRate,double *errl2)
{

for(int k=0;k<500;k++)
			for(int j=0;j<500;j++)
			{
				node2weights[k][j] = node2weights[k][j] + errRate*(errl2[k]*forpassl1[j]);
			}

}
__global__ void feed_forward_loop12(int training_sample,double node1weights[500][9],double errRate,double *errl1,int attributes[683][9])
{

for(int k=0;k<500;k++)
			for(int j=0;j<9;j++)
			{
				node1weights[k][j] = node1weights[k][j] + errRate*(errl1[k]*attributes[training_sample][j]);
			}



}


int main()
{
printf("Preparing Data For Training\n");
const char filename[] = "cancer_attributes.csv";

int **attributes =(int **)hipMallocManaged()


double *forpassl1,*forpassl2,*forpassl3,*forpassout;
hipMallocManaged(&forpassl1,500*sizeof(int));
hipMallocManaged(&forpassl2,500*sizeof(int));
hipMallocManaged(&forpassl3,500*sizeof(int));
hipMallocManaged(&forpassout,2*sizeof(int));

double *errl1,*errl2,*errl3,*errout;
hipMallocManaged(&errl1,500*sizeof(double));
hipMallocManaged(&errl2,500*sizeof(double));
hipMallocManaged(&errl3,500*sizeof(double));
hipMallocManaged(&errout,2*sizeof(double));

FILE *file = fopen(filename, "r");
   if ( file )
   { size_t i, j;char buffer[BUFSIZ], *ptr;
      for ( i = 0; fgets(buffer, sizeof buffer, file); ++i )
      {for ( j = 0, ptr = buffer; j < ARRAYSIZE(*attributes); ++j, ++ptr )
         {attributes[i][j] = (int)strtol(ptr, &ptr, 10);}
      }
      fclose(file);
  }


for (int i =0;i<683;i++){
  for(int j=0;j<9;j++)
  {printf("%d ",attributes[i][j]);}
  printf("\n");
  }


const char cancer_file_name[] = "cancer_classes.csv";


   FILE *cancer_file = fopen(cancer_file_name, "r");
   if ( cancer_file )
   { 
      size_t i1, j1;char buffer1[BUFSIZ], *ptr1;
      for ( i1 = 0; fgets(buffer1, sizeof buffer1, cancer_file); ++i1 )
      {for ( j1 = 0, ptr1 = buffer1; j1 < ARRAYSIZE(*classes); ++j1, ++ptr1 )
         {classes[i1][j1] = (int)strtol(ptr1, &ptr1, 10);}
      }
      fclose(cancer_file);}

for (int i =0;i<683;i++){
  for(int j=0;j<2;j++)
  {printf("%d ",classes[i][j]);}
  printf("\n");
 }


printf("Training Begins\n");


printf("Initializing Weights of layer 1\n");
for(int i=0;i<500;i++)for(int j=0;j<9;j++)node1weights[i][j]=0.25;
printf("Initializing Weights of layer 2\n");
for(int i=0;i<500;i++)for(int j=0;j<500;j++)node2weights[i][j]=0.25;
printf("Initializing Weights of layer 3\n");
for(int i=0;i<500;i++)for(int j=0;j<500;j++)node3weights[i][j]=0.25;
printf("initializing Weights for outer layer\n");
for(int i=0;i<2;i++)for(int j=0;j<500;j++)outputlayer[i][j]=0.25;




printf("Starting Training\n");



for(int training_sample = 0;training_sample<683;training_sample++)
{



feed_forward_loop1<<<1000,256>>>(training_sample,forpassl1,node1weights,attributes);

hipDeviceSynchronize();

feed_forward_loop2<<<1000,256>>>(training_sample,forpassl2,node2weights,forpassl1);

hipDeviceSynchronize();

feed_forward_loop3<<<1000,256>>>(training_sample,forpassl3,node3weights,forpassl2);

hipDeviceSynchronize();

feed_forward_loop4<<<1000,256>>>(forpassout,outputlayer,forpassl3);
hipDeviceSynchronize();

feed_forward_loop5<<<1000,256>>>(training_sample,errout,forpassout,classes);
hipDeviceSynchronize();

feed_forward_loop6<<<1000,256>>>(errout,outputlayer,errl3,forpassl3);
hipDeviceSynchronize();

feed_forward_loop7<<<1000,256>>>(errl3,node3weights,forpassl2,errl2);
hipDeviceSynchronize();

feed_forward_loop8<<<1000,256>>>(errl2,node2weights,forpassl1,errl1);
hipDeviceSynchronize();


feed_forward_loop9<<<1000,256>>>(outputlayer,errRate,errout,forpassl3);
hipDeviceSynchronize();

feed_forward_loop10<<<1000,256>>>(node3weights,forpassl2,errRate,errl3);
hipDeviceSynchronize();

feed_forward_loop11<<<1000,256>>>(node2weights,forpassl1,errRate,errl2);
hipDeviceSynchronize();

feed_forward_loop12<<<1000,256>>>(training_sample,node1weights,errRate,errl1,attributes);
hipDeviceSynchronize();
}
printf("Training Complete\n");


}
